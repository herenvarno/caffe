#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
////////////////////////////////////////////////////////////////////////////////
// MODIFICATION BEGIN
////////////////////////////////////////////////////////////////////////////////

template <>
void caffe_gpu_gemm_mod<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C, const int offset) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, offset));
}

template <>
void caffe_gpu_gemm_mod<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C, const int offset) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, offset));
}
////////////////////////////////////////////////////////////////////////////////
// MODIFICATION END
////////////////////////////////////////////////////////////////////////////////

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

....

}  // namespace caffe
